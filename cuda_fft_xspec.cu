#include "hip/hip_runtime.h"
//	cuda_fft_xspec.c : FFT using CuFFT
//
//	Author : Seiji Kameno
//	Created: 2012/12/6
//
#include "shm_k5data.inc"
#include "k5dict.inc"
#include <hip/hip_runtime.h>
#include <cutil.h>
#include <hipfft/hipfft.h>
#define	NFFT		262144
#define	NSEG		256

__device__ float2 complexMult(float2 a, float2 b)		// output a x b
{
	return make_float2( a.x* b.x - a.y* b.y, a.x* b.y + a.y* b.x );
}

__device__ float2 complexMultConj(float2 a, float2 b)		// output a x b*
{
	return make_float2( a.x* b.x + a.y* b.y,  a.y* b.x - a.x* b.y );
}

__device__ float complexMod( float2 a )				// output |a|^2
{
	return  a.x* a.x + a.y* a.y 
}

__global__ void complexMultConjVec(		// calculate a x b*
	float2	*vec_in_a,			// Input vector
	float2	*vec_in_b,			// Input vector
	float2	*vec_out_c,			// Output vector
	int		length)				// Vector length
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;
	if((tid >= 0) && (tid < length)){
		vec_out_c[tid] = complexMultConj(vec_in_a[tid], vec_in_b[tid]);
	}
}

__global__ void complexPowerVec(		// calculate a x a*
	float2	*vec_in,		// Input vector
	float	*vec_out,
	int		length)
{
	int tid = blockIdx.x* blockDim.x + threadIdx.x;
	if((tid >= 0) && (tid < length)){
		vec_out[tid] = complexMod(vec_in_a[tid]);
	}
}


main(
	int		argc,			// Number of Arguments
	char	**argv )		// Pointer to Arguments
{
	int		shrd_param_id;				// Shared Memory ID
	struct	SHM_PARAM	*param_ptr;		// Pointer to the Shared Param
	struct	sembuf		sops;			// Semaphore for data access
	float	*segdata_ptr;				// Pointer to sampled data

	hipfftHandle		cufft_plan;
	hipfftComplex	*cuSpec_data;
	hipfftReal		*cuReal_data;
/*
-------------------------------------------- Prepare for CuFFT
*/
	hipMalloc( (void **)&cuReal_data, SEGDATA_SIZE/2 );
	hipMalloc( (void **)&cuSpec_data, SEGDATA_SIZE/2 );
	if(hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda Error : Failed to allocate memory.\n"); return(-1);
	}

	if(hipfftPlan1d(&cufft_plan, NFFT, HIPFFT_R2C, NSEG) != HIPFFT_SUCCESS){
		fprintf(stderr, "Cuda Error : Failed to create plan.\n"); return(-1);
	}
	
/*
-------------------------------------------- Access to the SHARED MEMORY
*/
	shrd_param_id = shmget( SHM_PARAM_KEY, sizeof(struct SHM_PARAM), 0444);
	param_ptr = (struct SHM_PARAM *)shmat(shrd_param_id, NULL, 0);
	segdata_ptr = (float *)shmat(param_ptr->shrd_seg_id, NULL, SHM_RDONLY);
/*
-------------------------------------------- K5 Header and Data
*/
	setvbuf(stdout, (char *)NULL, _IONBF, 0);   // Disable stdout cache
	while(param_ptr->validity & ACTIVE){
		if( param_ptr->validity & (FINISH + ABSFIN) ){  break; }

		printf("FFT status EW\r");
		sops.sem_num = (ushort)8; sops.sem_op = (short)-4; sops.sem_flg = (short)0;
        semop( param_ptr->sem_data_id, &sops, 1);

		printf("FFT status ES\r");
		hipMemcpy( cuReal_data, segdata_ptr, SEGDATA_SIZE/2, hipMemcpyHostToDevice);
		printf("FFT status EF\r");
		hipfftExecR2C(cufft_plan, cuReal_data, cuSpec_data);

		printf("FFT status OW\r");
		segdata_ptr += MAX_IF* MAX_seg_len* MAX_seg_sec / 2;
		sops.sem_num = (ushort)9; sops.sem_op = (short)-4; sops.sem_flg = (short)0;
        semop( param_ptr->sem_data_id, &sops, 1);
		printf("FFT status OS\r");
		hipMemcpy( cuReal_data, segdata_ptr, SEGDATA_SIZE/2, hipMemcpyHostToDevice);
		segdata_ptr -= MAX_IF* MAX_seg_len* MAX_seg_sec / 2;
		printf("FFT status OF\r");

		hipfftExecR2C(cufft_plan, cuReal_data, cuSpec_data);
	}
/*
-------------------------------------------- RELEASE the SHM
*/
	hipfftDestroy(cufft_plan);
	hipFree(&cuReal_data);
	hipFree(&cuSpec_data);
    return(0);
}
