#include "hip/hip_runtime.h"
//	cuda_fft_xspec.c : FFT using CuFFT
//
//	Author : Seiji Kameno
//	Created: 2012/12/6
//
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <string.h>
#include <math.h>
#include </usr/local/cuda-5.5/samples/common/inc/timer.h>
#include "cuda_polaris.inc"
#define	PARTNUM 2
#define SCALEFACT 2.0/(NFFT* NsegSec* PARTNUM)

extern int gaussBit(int, unsigned int *, double *, double *);
extern int k5utc(unsigned char *,	struct SHM_PARAM *);
extern int fileRecOpen(struct SHM_PARAM *, int, int, char *, char *, FILE **);
extern int bitDist4(int, char *, unsigned int *);
extern int bitDist8(int, char *, unsigned int *);

main(
	int		argc,			// Number of Arguments
	char	**argv )		// Pointer to Arguments
{
	int		shrd_param_id;				// Shared Memory ID
	int		index;						// General Index
	int		part_index;					// First and Last Part
	int		seg_index;					// Index for Segment
	int		offset[1024];				// Segment offset position
	int		sod = 0;					// Seconds of Day
	int		nlevel;						// Number of quantized levels (2/4/16/256)
	unsigned char		*k5head_ptr;	// Pointer to the K5 header
	struct	SHM_PARAM	*param_ptr;		// Pointer to the Shared Param
	struct	sembuf		sops;			// Semaphore for data access
	char	*k5data_ptr;				// Pointer to shared K5 data
	float	*xspec_ptr;					// Pointer to 1-sec-integrated Power Spectrum
	FILE	*file_ptr[6];				// File Pointer to write
	FILE	*power_ptr[4];				// Power File Pointer to write
	char	fname_pre[16];
	unsigned int		bitDist[1024];
	double	param[2], param_err[2];		// Gaussian parameters derived from bit distribution

	dim3			Dg, Db(512,1, 1);	// Grid and Block size
	unsigned char	*cuk5data_ptr;		// Pointer to K5 data
	hipfftHandle		cufft_plan;			// 1-D FFT Plan, to be used in cufft
	hipfftReal		*cuRealData;		// Time-beased data before FFT, every IF, every segment
	hipfftComplex	*cuSpecData;		// FFTed spectrum, every IF, every segment
	float			*cuPowerSpec;		// (autocorrelation) Power Spectrum
	float2			*cuXSpec;

//------------------------------------------ Access to the SHARED MEMORY
	shrd_param_id = shmget( SHM_PARAM_KEY, sizeof(struct SHM_PARAM), 0444);
	param_ptr  = (struct SHM_PARAM *)shmat(shrd_param_id, NULL, 0);
	k5data_ptr = (char *)shmat(param_ptr->shrd_k5data_id, NULL, SHM_RDONLY);
	xspec_ptr  = (float *)shmat(param_ptr->shrd_xspec_id, NULL, 0);
	k5head_ptr = (unsigned char *)shmat(param_ptr->shrd_k5head_id, NULL, SHM_RDONLY);
//------------------------------------------ Prepare for CuFFT
	hipMalloc( (void **)&cuk5data_ptr, MAX_SAMPLE_BUF);
	hipMalloc( (void **)&cuRealData, Nif* NsegSec2* NFFT * sizeof(hipfftReal) );
	hipMalloc( (void **)&cuSpecData, Nif* NsegSec2* NFFTC* sizeof(hipfftComplex) );
	hipMalloc( (void **)&cuPowerSpec, Nif* NFFT2* sizeof(float));
	hipMalloc( (void **)&cuXSpec, 2* NFFT2* sizeof(float2));

	if(hipGetLastError() != hipSuccess){
		fprintf(stderr, "Cuda Error : Failed to allocate memory.\n"); return(-1);
	}

	if(hipfftPlan1d(&cufft_plan, NFFT, HIPFFT_R2C, Nif* NsegSec2 ) != HIPFFT_SUCCESS){
		fprintf(stderr, "Cuda Error : Failed to create plan.\n"); return(-1);
	}
	printf("NsegSec2 = %d\n", NsegSec2);
//------------------------------------------ Parameters for S-part format
	for(seg_index = 0; seg_index < NsegSec2; seg_index ++){
		offset[seg_index] = seg_index* (param_ptr->fsample/2 - param_ptr->segLen) / (NsegSec2 - 1);
	}
	for(seg_index = NsegSec2; seg_index < NsegSec; seg_index ++){
		offset[seg_index] = (seg_index - 1)* (param_ptr->fsample/2 - param_ptr->segLen/2) / (NsegSec2 - 1) ;
	}
	nlevel = 0x01<<(param_ptr->qbit);		// Number of levels = 2^qbit
//------------------------------------------ K5 Header and Data
	param_ptr->current_rec = 0;
	setvbuf(stdout, (char *)NULL, _IONBF, 0);   // Disable stdout cache
	while(param_ptr->validity & ACTIVE){
		if( param_ptr->validity & (FINISH + ABSFIN) ){  break; }
		hipMemset( cuPowerSpec, 0, Nif* NFFT2* sizeof(float));		// Clear Power Spec
		hipMemset( cuXSpec, 0, 2* NFFT2* sizeof(float2));		// Clear Power Spec

		//-------- UTC in the K5 header
		while(k5utc(k5head_ptr, param_ptr) == 0){	usleep(100000);}

		//-------- Open output files
		if(param_ptr->current_rec == 0){
			sprintf(fname_pre, "%04d%03d%02d%02d%02d", param_ptr->year, param_ptr->doy, param_ptr->hour, param_ptr->min, param_ptr->sec );
			for(index=0; index<Nif; index++){
				fileRecOpen(param_ptr, index, (A00_REC << index), fname_pre, "A", file_ptr);		// Autocorr
				fileRecOpen(param_ptr, index, (P00_REC << index), fname_pre, "P", power_ptr);		// Bitpower
			}
			for(index=0; index<Nif/2; index++){
				fileRecOpen(param_ptr, index, (C00_REC << index), fname_pre, "C", &file_ptr[Nif]);	// Crosscorr
			}
		}

		memset(bitDist, 0, sizeof(bitDist));
		for(part_index=0; part_index<PARTNUM; part_index ++){
			//-------- Wait for the first half in the S-part
			sops.sem_num = (ushort)(4* part_index); sops.sem_op = (short)-1; sops.sem_flg = (short)0;
			semop( param_ptr->sem_data_id, &sops, 1);

			//-------- Move K5-sample data onto GPU memory
			// StartTimer();
			hipMemcpy( &cuk5data_ptr[part_index* HALFBUF], &k5data_ptr[part_index* HALFBUF], HALFBUF, hipMemcpyHostToDevice);

			//-------- Segment Format and Bit Distribution
			hipDeviceSynchronize();
			Dg.x=NFFT/512; Dg.y=1; Dg.z=1;
			if( nlevel == 256){
				for(index=0; index < NsegSec2; index ++){
					seg_index = part_index* NsegSec2 + index;
					segform8bit<<<Dg, Db>>>( &cuk5data_ptr[4* offset[seg_index]], &cuRealData[index* Nif* NFFT], NFFT);
				}
				bitDist8( HALFBUF/2, &k5data_ptr[part_index* HALFBUF], bitDist);
			} else{
				for(index=0; index < NsegSec2; index ++){
					seg_index = part_index* NsegSec2 + index;
					segform4bit<<<Dg, Db>>>( &cuk5data_ptr[2* offset[seg_index]], &cuRealData[index* Nif* NFFT], NFFT);
				}
				bitDist4( HALFBUF, &k5data_ptr[part_index* HALFBUF], bitDist);
			}

			//-------- FFT Real -> Complex spectrum
			hipDeviceSynchronize();
			hipfftExecR2C(cufft_plan, cuRealData, cuSpecData);			// FFT Time -> Freq
			hipDeviceSynchronize();

			//---- Auto Corr
			Dg.x= NFFTC/512; Dg.y=1; Dg.z=1;
			for(seg_index=0; seg_index<NsegSec2; seg_index++){
				for(index=0; index<Nif; index++){
					accumPowerSpec<<<Dg, Db>>>( &cuSpecData[(seg_index* Nif + index)* NFFTC], &cuPowerSpec[index* NFFT2],  NFFT2);
				}
			}
			//---- Cross Corr
			for(seg_index=0; seg_index<NsegSec2; seg_index++){
				accumCrossSpec<<<Dg, Db>>>( &cuSpecData[(seg_index* Nif)* NFFTC], &cuSpecData[(seg_index* Nif + 2)* NFFTC], cuXSpec, NFFT2);
				accumCrossSpec<<<Dg, Db>>>( &cuSpecData[(seg_index* Nif + 1)*NFFTC], &cuSpecData[(seg_index* Nif + 3)*NFFTC], &cuXSpec[NFFT2], NFFT2);
			}
			// printf("%lf [msec]\n", GetTimer());
		}	// End of part loop
		Dg.x = Nif* NFFT2/512; Dg.y=1; Dg.z=1;
		scalePowerSpec<<<Dg, Db>>>(cuPowerSpec, SCALEFACT, Nif* NFFT2);
		scaleCrossSpec<<<Dg, Db>>>(cuXSpec, SCALEFACT, 2* NFFT2);

		//-------- Dump cross spectra to shared memory
		hipMemcpy(xspec_ptr, cuPowerSpec, Nif* NFFT2* sizeof(float), hipMemcpyDeviceToHost);
		for(index=0; index<Nif; index++){
			if(file_ptr[index] != NULL){fwrite(&xspec_ptr[index* NFFT2], sizeof(float), NFFT2, file_ptr[index]);}	// Save Pspec
			if(power_ptr[index] != NULL){fwrite(&bitDist[index* 16], sizeof(int), 16, power_ptr[index]);}			// Save Bitdist
			//-------- Total Power calculation
			gaussBit( nlevel, &bitDist[nlevel* index], param, param_err );
			param_ptr->power[index] = 1.0/(param[0]* param[0]);
		}
		hipMemcpy(&xspec_ptr[4* NFFT2], cuXSpec, 2* NFFT2* sizeof(float2), hipMemcpyDeviceToHost);
		for(index=0; index<Nif/2; index++){
			if(file_ptr[Nif + index] != NULL){
				fwrite(&xspec_ptr[(Nif + index * 2)* NFFT2], sizeof(float2), NFFT2, file_ptr[Nif + index]);	// Save Xspec
			}
		}

		//-------- Refresh output data file
		if(param_ptr->current_rec == MAX_FILE_REC - 1){
			for(index=0; index<Nif+2; index++){ if( file_ptr[index] != NULL){	fclose(file_ptr[index]);} }
			for(index=0; index<Nif; index++){ if( power_ptr[index] != NULL){	fclose(power_ptr[index]);} }
			param_ptr->current_rec = 0;
		} else { param_ptr->current_rec ++; }

		sops.sem_num = (ushort)SEM_FX; sops.sem_op = (short)1; sops.sem_flg = (short)0; semop( param_ptr->sem_data_id, &sops, 1);
		sops.sem_num = (ushort)SEM_POWER; sops.sem_op = (short)1; sops.sem_flg = (short)0; semop( param_ptr->sem_data_id, &sops, 1);
		printf("%04d %03d SOD=%d UT=%02d:%02d:%02d Rec %d / %d -- Succeeded.\n",
			param_ptr->year, param_ptr->doy, sod, param_ptr->hour, param_ptr->min, param_ptr->sec, param_ptr->current_rec, param_ptr->integ_rec);
	}	// End of 1-sec loop
/*
-------------------------------------------- RELEASE the SHM
*/
	for(index=0; index<Nif+2; index++){ if( file_ptr[index] != NULL){	fclose(file_ptr[index]);} }
	for(index=0; index<Nif; index++){ if( power_ptr[index] != NULL){	fclose(power_ptr[index]);} }
	hipfftDestroy(cufft_plan);
	hipFree(cuk5data_ptr); hipFree(cuRealData); hipFree(cuSpecData); hipFree(cuPowerSpec), hipFree(cuXSpec);

    return(0);
}

